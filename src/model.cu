#include "hip/hip_runtime.h"
#include <NvInferRuntime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void upcast_uint16_to_int64_kernel() {

}

extern "C" void upcast_uint16_to_int64(hipStream_t stream) {
	upcast_uint16_to_int64_kernel<<<1, 1, 0, stream>>>();
}


class MyLogger : public nvinfer1::ILogger {
        void log(nvinfer1::ILogger::Severity severity, nvinfer1::AsciiChar const* msg) noexcept override {
		printf("%s\n", msg);
	}
} logger;

struct EngineInfo {
	unsigned char* data;
	size_t size;
};

// TODO: handle error more gracefully
EngineInfo read_engine_file(char* engine_file) {
    FILE* file = fopen(engine_file, "rb");

    fseek(file, 0, SEEK_END);
    size_t size = ftell(file);
    rewind(file);

    unsigned char* data = (unsigned char*)malloc(size);
    fread(data, 1, size, file);

    fclose(file);
    EngineInfo info;
    info.data = data;
    info.size = size;
    return info;
}

extern "C" nvinfer1::IRuntime* create_runtime() {
	return nvinfer1::createInferRuntime(logger);
}

extern "C" void destroy_runtime(nvinfer1::IRuntime* rt) {
	delete rt;
}

extern "C" nvinfer1::ICudaEngine* create_engine(nvinfer1::IRuntime* rt, char* path) {
	EngineInfo info = read_engine_file(path);
	nvinfer1::ICudaEngine* eng = rt->deserializeCudaEngine(info.data, info.size);
	free(info.data);
	return eng;
}

extern "C" void destroy_engine(nvinfer1::ICudaEngine* eng) {
	delete eng;
}

extern "C" nvinfer1::IExecutionContext* create_execution_context(nvinfer1::ICudaEngine* eng) {
	return eng->createExecutionContext();
}

extern "C" void destroy_execution_context(nvinfer1::IExecutionContext* ctx) {
	delete ctx;
}

extern "C" void set_tensor_shape(nvinfer1::IExecutionContext* ctx, int32_t batch, int32_t size) {
	auto dims = nvinfer1::Dims64();
	dims.nbDims = 2;
	dims.d[0] = batch;
	dims.d[1] = size;
	ctx->setInputShape("input_ids", dims);
	ctx->setInputShape("attention_mask", dims);
}

extern "C" bool enqueue(nvinfer1::IExecutionContext* ctx, hipStream_t stream) {
        return ctx->enqueueV3(stream);
}

extern "C" void set_device_memory(nvinfer1::IExecutionContext* ctx, void* address, int64_t size) {
	ctx->setDeviceMemoryV2(address, size);
}

extern "C" int64_t get_device_memory_size(nvinfer1::ICudaEngine* eng, int32_t profile) {
	return eng->getDeviceMemorySizeForProfileV2(profile);
}

